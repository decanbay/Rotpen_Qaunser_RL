#include "hip/hip_runtime.h"
/* Include files */

#include "Non_linear_simulation_sfun.h"
#include "c2_Non_linear_simulation.h"
#include "MWCudaDimUtility.hpp"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include "MWCUBLASUtils.hpp"

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void initialize_params_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void enable_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void disable_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void c2_do_animation_call_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void ext_mode_exec_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void set_sim_state_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance, const mxArray *c2_st);
static void sf_gateway_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void mdl_start_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void mdl_terminate_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void initSimStructsc2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance);
static void c2_eML_blk_kernel(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, real_T c2_b_signals[3], real_T c2_b_iterationCounter, real_T
  c2_b_armPartPatchHandle, real_T c2_b_pendPartPatchHandle, real_T
  c2_b_drivePartPatchHandle, real_T c2_b_thetaPlotHandle, real_T
  c2_b_alphaPlotHandle, real_T c2_b_frameRate, real_T c2_b_armPartVertices[88224],
  real_T c2_b_drivePartVertices[14976], real_T c2_b_pendPartVertices[26064]);
static uint8_T c2_emlrt_marshallIn(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_b_is_active_c2_Non_linear_simulation, const
  char_T *c2_identifier);
static uint8_T c2_b_emlrt_marshallIn(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static void c2_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                   const mxArray *c2_input0, const mxArray *c2_input1, const
                   mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                   *c2_input4);
static void c2_b_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4);
static void c2_c_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4, const mxArray *c2_input5, const mxArray
                     *c2_input6);
static void c2_d_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4, const mxArray *c2_input5, const mxArray
                     *c2_input6);
static void c2_e_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4, const mxArray *c2_input5, const mxArray
                     *c2_input6);
static const mxArray *c2_get(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1);
static const mxArray *c2_horzcat(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1);
static const mxArray *c2_b_get(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1);
static const mxArray *c2_b_horzcat(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1);
static void c2_f_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4);
static const mxArray *c2_c_get(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1);
static const mxArray *c2_c_horzcat(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1);
static const mxArray *c2_d_get(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1);
static const mxArray *c2_d_horzcat(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1);
static void c2_g_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4);
static __global__ void c2_eML_blk_kernel_kernel1(const real_T c2_sy, const
  real_T c2_cy, const real_T c2_b_sy, const real_T c2_b_cy, real_T
  c2_BaseToDrive[16], real_T c2_BaseToArm[16]);
static __global__ void c2_eML_blk_kernel_kernel2(const int8_T c2_b_dv1[4], const
  real_T c2_dv[4], real_T c2_BaseToDrive[16], real_T c2_BaseToArm[16]);
static __global__ void c2_eML_blk_kernel_kernel3(real_T
  c2_armPart_vertices_G_frame[88224]);
static __global__ void c2_eML_blk_kernel_kernel4(real_T c2_b_Zt[22056], real_T
  c2_armPart_vertices_G_frame[22056], real_T c2_b_Yt[22056], real_T
  c2_b_armPart_vertices_G_frame[22056], real_T c2_c_Xt[22056], real_T
  c2_c_armPart_vertices_G_frame[88224], real_T c2_d_armPart_vertices_G_frame
  [22056]);
static __global__ void c2_eML_blk_kernel_kernel5(const real_T c2_sr, const
  real_T c2_cr, real_T c2_dv[16]);
static __global__ void c2_eML_blk_kernel_kernel6(const int8_T c2_b_dv1[4],
  real_T c2_dv[16]);
static __global__ void c2_eML_blk_kernel_kernel7(const real_T c2_dv[16], real_T
  c2_BaseToArm[16], real_T c2_y[16]);
static __global__ void c2_eML_blk_kernel_kernel8(real_T
  c2_pendPart_vertices_G_frame[26064]);
static __global__ void c2_eML_blk_kernel_kernel9(real_T c2_b_Zt[6516], real_T
  c2_pendPart_vertices_G_frame[6516], real_T c2_b_Yt[6516], real_T
  c2_b_pendPart_vertices_G_frame[6516], real_T c2_c_Xt[6516], real_T
  c2_c_pendPart_vertices_G_frame[26064], real_T c2_d_pendPart_vertices_G_frame
  [6516]);
static __global__ void c2_eML_blk_kernel_kernel10(real_T
  c2_drivePart_vertices_G_frame[14976]);
static __global__ void c2_eML_blk_kernel_kernel11(real_T c2_b_Zt[3744], real_T
  c2_drivePart_vertices_G_frame[3744], real_T c2_b_Yt[3744], real_T
  c2_b_drivePart_vertices_G_frame[3744], real_T c2_c_Xt[3744], real_T
  c2_c_drivePart_vertices_G_frame[14976], real_T
  c2_d_drivePart_vertices_G_frame[3744]);
static void init_dsm_address_info(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c2_fEmlrtCtx,
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c2_is_active_c2_Non_linear_simulation = 0U;
  cublasEnsureInitialization(HIPBLAS_POINTER_MODE_DEVICE);
  hipGetLastError();
  hipMalloc(&chartInstance->c2_d_gpu_armPart_vertices_G_frame, 176448UL);
  hipMalloc(&chartInstance->c2_gpu_BaseToDrive, 128UL);
  hipMalloc(&chartInstance->c2_b_gpu_sy, 8UL);
  hipMalloc(&chartInstance->c2_gpu_BaseToArm, 128UL);
  hipMalloc(&chartInstance->c2_gpu_drivePartVertices, 119808UL);
  hipMalloc(&chartInstance->c2_gpu_drivePart_vertices_G_frame, 119808UL);
  hipMalloc(&chartInstance->c2_gpu_Yt, 52128UL);
  hipMalloc(&chartInstance->c2_b_gpu_alpha1, 8UL);
  hipMalloc(&chartInstance->c2_b_gpu_armPart_vertices_G_frame, 176448UL);
  hipMalloc(&chartInstance->c2_gpu_alpha1, 8UL);
  hipMalloc(&chartInstance->c2_gpu_Zt, 29952UL);
  hipMalloc(&chartInstance->c2_gpu_armPart_vertices_G_frame, 705792UL);
  hipMalloc(&chartInstance->c2_gpu_sr, 8UL);
  hipMalloc(&chartInstance->c2_c_gpu_pendPart_vertices_G_frame, 52128UL);
  hipMalloc(&chartInstance->c2_gpu_beta1, 8UL);
  hipMalloc(&chartInstance->c2_c_gpu_Zt, 176448UL);
  hipMalloc(&chartInstance->c2_d_gpu_pendPart_vertices_G_frame, 52128UL);
  hipMalloc(&chartInstance->c2_b_gpu_beta1, 8UL);
  hipMalloc(&chartInstance->c2_gpu_Xt, 176448UL);
  hipMalloc(&chartInstance->c2_b_gpu_Xt, 52128UL);
  hipMalloc(&chartInstance->c2_c_gpu_alpha1, 8UL);
  hipMalloc(&chartInstance->c2_b_gpu_dv, 128UL);
  hipMalloc(&chartInstance->c2_gpu_dv1, 4UL);
  hipMalloc(&chartInstance->c2_gpu_cy, 8UL);
  hipMalloc(&chartInstance->c2_b_gpu_Zt, 52128UL);
  hipMalloc(&chartInstance->c2_d_gpu_drivePart_vertices_G_frame, 29952UL);
  hipMalloc(&chartInstance->c2_c_gpu_armPart_vertices_G_frame, 176448UL);
  hipMalloc(&chartInstance->c2_b_gpu_Yt, 29952UL);
  hipMalloc(&chartInstance->c2_b_gpu_cy, 8UL);
  hipMalloc(&chartInstance->c2_c_gpu_Yt, 176448UL);
  hipMalloc(&chartInstance->c2_b_gpu_pendPart_vertices_G_frame, 208512UL);
  hipMalloc(&chartInstance->c2_gpu_sy, 8UL);
  hipMalloc(&chartInstance->c2_gpu_dv, 32UL);
  hipMalloc(&chartInstance->c2_gpu_pendPart_vertices_G_frame, 52128UL);
  hipMalloc(&chartInstance->c2_c_gpu_Xt, 29952UL);
  hipMalloc(&chartInstance->c2_b_gpu_drivePart_vertices_G_frame, 29952UL);
  hipMalloc(&chartInstance->c2_c_gpu_drivePart_vertices_G_frame, 29952UL);
  hipMalloc(&chartInstance->c2_gpu_armPartVertices, 705792UL);
  hipMalloc(&chartInstance->c2_c_gpu_beta1, 8UL);
  hipMalloc(&chartInstance->c2_gpu_y, 128UL);
  hipMalloc(&chartInstance->c2_gpu_cr, 8UL);
  hipMalloc(&chartInstance->c2_gpu_pendPartVertices, 208512UL);
}

static void initialize_params_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
}

static void enable_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c2_do_animation_call_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
  const mxArray *c2_b_y = NULL;
  const mxArray *c2_st = NULL;
  const mxArray *c2_y = NULL;
  c2_st = NULL;
  c2_y = NULL;
  sf_mex_assign(&c2_y, sf_mex_createcellmatrix(1, 1), false);
  c2_b_y = NULL;
  sf_mex_assign(&c2_b_y, sf_mex_create("y",
    &chartInstance->c2_is_active_c2_Non_linear_simulation, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c2_y, 0, c2_b_y);
  sf_mex_assign(&c2_st, c2_y, false);
  return c2_st;
}

static void set_sim_state_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance, const mxArray *c2_st)
{
  const mxArray *c2_u;
  c2_u = sf_mex_dup(c2_st);
  chartInstance->c2_is_active_c2_Non_linear_simulation = c2_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 0)),
     "is_active_c2_Non_linear_simulation");
  sf_mex_destroy(&c2_u);
  sf_mex_destroy(&c2_st);
}

static void sf_gateway_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
  real_T c2_dv[3];
  int32_T c2_i;
  int32_T c2_i1;
  int32_T c2_i2;
  int32_T c2_i3;
  chartInstance->c2_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c2_i = 0; c2_i < 3; c2_i++) {
    c2_dv[c2_i] = (*chartInstance->c2_signals)[c2_i];
  }

  for (c2_i1 = 0; c2_i1 < 88224; c2_i1++) {
    chartInstance->c2_dv1[c2_i1] = (*chartInstance->c2_armPartVertices)[c2_i1];
  }

  for (c2_i2 = 0; c2_i2 < 14976; c2_i2++) {
    chartInstance->c2_dv2[c2_i2] = (*chartInstance->c2_drivePartVertices)[c2_i2];
  }

  for (c2_i3 = 0; c2_i3 < 26064; c2_i3++) {
    chartInstance->c2_dv3[c2_i3] = (*chartInstance->c2_pendPartVertices)[c2_i3];
  }

  c2_eML_blk_kernel(chartInstance, c2_dv, *chartInstance->c2_iterationCounter,
                    *chartInstance->c2_armPartPatchHandle,
                    *chartInstance->c2_pendPartPatchHandle,
                    *chartInstance->c2_drivePartPatchHandle,
                    *chartInstance->c2_thetaPlotHandle,
                    *chartInstance->c2_alphaPlotHandle,
                    *chartInstance->c2_frameRate, chartInstance->c2_dv1,
                    chartInstance->c2_dv2, chartInstance->c2_dv3);
  c2_do_animation_call_c2_Non_linear_simulation(chartInstance);
}

static void mdl_start_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
  hipError_t c2_errCode;
  hipFree(*chartInstance->c2_c_gpu_pendPart_vertices_G_frame);
  hipFree(*chartInstance->c2_b_gpu_Yt);
  hipFree(*chartInstance->c2_c_gpu_Zt);
  hipFree(*chartInstance->c2_b_gpu_dv);
  hipFree(chartInstance->c2_c_gpu_alpha1);
  hipFree(*chartInstance->c2_gpu_drivePart_vertices_G_frame);
  hipFree(*chartInstance->c2_c_gpu_Yt);
  hipFree(*chartInstance->c2_gpu_Xt);
  hipFree(*chartInstance->c2_b_gpu_pendPart_vertices_G_frame);
  hipFree(*chartInstance->c2_gpu_armPart_vertices_G_frame);
  hipFree(*chartInstance->c2_gpu_BaseToDrive);
  hipFree(*chartInstance->c2_c_gpu_armPart_vertices_G_frame);
  hipFree(chartInstance->c2_gpu_sr);
  hipFree(chartInstance->c2_gpu_cy);
  hipFree(chartInstance->c2_gpu_alpha1);
  hipFree(chartInstance->c2_c_gpu_beta1);
  hipFree(*chartInstance->c2_gpu_Yt);
  hipFree(chartInstance->c2_b_gpu_cy);
  hipFree(*chartInstance->c2_gpu_drivePartVertices);
  hipFree(chartInstance->c2_b_gpu_sy);
  hipFree(*chartInstance->c2_c_gpu_drivePart_vertices_G_frame);
  hipFree(*chartInstance->c2_b_gpu_armPart_vertices_G_frame);
  hipFree(chartInstance->c2_gpu_beta1);
  hipFree(chartInstance->c2_gpu_sy);
  hipFree(*chartInstance->c2_gpu_armPartVertices);
  hipFree(*chartInstance->c2_b_gpu_drivePart_vertices_G_frame);
  hipFree(*chartInstance->c2_b_gpu_Zt);
  hipFree(*chartInstance->c2_d_gpu_armPart_vertices_G_frame);
  hipFree(chartInstance->c2_b_gpu_alpha1);
  hipFree(*chartInstance->c2_d_gpu_drivePart_vertices_G_frame);
  hipFree(*chartInstance->c2_c_gpu_Xt);
  hipFree(*chartInstance->c2_b_gpu_Xt);
  hipFree(chartInstance->c2_gpu_cr);
  hipFree(*chartInstance->c2_gpu_pendPartVertices);
  hipFree(*chartInstance->c2_gpu_y);
  hipFree(*chartInstance->c2_gpu_pendPart_vertices_G_frame);
  hipFree(*chartInstance->c2_gpu_Zt);
  hipFree(*chartInstance->c2_gpu_BaseToArm);
  hipFree(*chartInstance->c2_d_gpu_pendPart_vertices_G_frame);
  hipFree(chartInstance->c2_b_gpu_beta1);
  hipFree(*chartInstance->c2_gpu_dv1);
  hipFree(*chartInstance->c2_gpu_dv);
  c2_errCode = hipGetLastError();
  if (c2_errCode != hipSuccess) {
    emlrtThinCUDAError(c2_errCode, hipGetErrorName(c2_errCode),
                       hipGetErrorString(c2_errCode), "SimGPUErrorChecks",
                       chartInstance->c2_fEmlrtCtx);
  }

  cublasEnsureDestruction();
}

static void mdl_setup_runtime_resources_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
}

static void initSimStructsc2_Non_linear_simulation
  (SFc2_Non_linear_simulationInstanceStruct *chartInstance)
{
}

const mxArray *sf_c2_Non_linear_simulation_get_eml_resolved_functions_info()
{
  const mxArray *c2_nameCaptureInfo = NULL;
  c2_nameCaptureInfo = NULL;
  sf_mex_assign(&c2_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c2_nameCaptureInfo;
}

static void c2_eML_blk_kernel(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, real_T c2_b_signals[3], real_T c2_b_iterationCounter, real_T
  c2_b_armPartPatchHandle, real_T c2_b_pendPartPatchHandle, real_T
  c2_b_drivePartPatchHandle, real_T c2_b_thetaPlotHandle, real_T
  c2_b_alphaPlotHandle, real_T c2_b_frameRate, real_T c2_b_armPartVertices[88224],
  real_T c2_b_drivePartVertices[14976], real_T c2_b_pendPartVertices[26064])
{
  static real_T c2_dv[4] = { -0.0, 0.0, 1.0, 0.12 };

  static char_T c2_cv[5] = { 'x', 'D', 'a', 't', 'a' };

  static char_T c2_cv1[5] = { 'y', 'D', 'a', 't', 'a' };

  static char_T c2_cv2[5] = { 'z', 'D', 'a', 't', 'a' };

  static char_T c2_cv3[5] = { 'x', 'd', 'a', 't', 'a' };

  static char_T c2_cv4[5] = { 'y', 'd', 'a', 't', 'a' };

  static int8_T c2_b_dv1[4] = { 0, 0, 0, 1 };

  const mxArray *c2_ab_y = NULL;
  const mxArray *c2_alphaAngles = NULL;
  const mxArray *c2_b_y = NULL;
  const mxArray *c2_bb_y = NULL;
  const mxArray *c2_c_y = NULL;
  const mxArray *c2_cb_y = NULL;
  const mxArray *c2_d_y = NULL;
  const mxArray *c2_db_y = NULL;
  const mxArray *c2_e_y = NULL;
  const mxArray *c2_eb_y = NULL;
  const mxArray *c2_f_y = NULL;
  const mxArray *c2_fb_y = NULL;
  const mxArray *c2_g_y = NULL;
  const mxArray *c2_gb_y = NULL;
  const mxArray *c2_h_y = NULL;
  const mxArray *c2_hb_y = NULL;
  const mxArray *c2_i_y = NULL;
  const mxArray *c2_ib_y = NULL;
  const mxArray *c2_j_y = NULL;
  const mxArray *c2_jb_y = NULL;
  const mxArray *c2_k_y = NULL;
  const mxArray *c2_kb_y = NULL;
  const mxArray *c2_l_y = NULL;
  const mxArray *c2_lb_y = NULL;
  const mxArray *c2_m_y = NULL;
  const mxArray *c2_mb_y = NULL;
  const mxArray *c2_n_y = NULL;
  const mxArray *c2_nb_y = NULL;
  const mxArray *c2_o_y = NULL;
  const mxArray *c2_ob_y = NULL;
  const mxArray *c2_p_y = NULL;
  const mxArray *c2_pb_y = NULL;
  const mxArray *c2_q_y = NULL;
  const mxArray *c2_qb_y = NULL;
  const mxArray *c2_r_y = NULL;
  const mxArray *c2_rb_y = NULL;
  const mxArray *c2_s_y = NULL;
  const mxArray *c2_sb_y = NULL;
  const mxArray *c2_t_y = NULL;
  const mxArray *c2_tb_y = NULL;
  const mxArray *c2_thetaAngles = NULL;
  const mxArray *c2_times = NULL;
  const mxArray *c2_u_y = NULL;
  const mxArray *c2_ub_y = NULL;
  const mxArray *c2_v_y = NULL;
  const mxArray *c2_vb_y = NULL;
  const mxArray *c2_w_y = NULL;
  const mxArray *c2_wb_y = NULL;
  const mxArray *c2_x_y = NULL;
  const mxArray *c2_xb_y = NULL;
  const mxArray *c2_y = NULL;
  const mxArray *c2_y_y = NULL;
  real_T c2_b_Yt[6516];
  real_T c2_b_Zt[6516];
  real_T c2_c_Xt[3744];
  real_T c2_c_Yt[3744];
  real_T c2_c_Zt[3744];
  real_T c2_b_dv[16];
  real_T c2_YAW;
  real_T c2_alpha1;
  real_T c2_b_alpha1;
  real_T c2_b_beta1;
  real_T c2_b_u;
  real_T c2_beta1;
  real_T c2_c_alpha1;
  real_T c2_c_beta1;
  real_T c2_c_u;
  real_T c2_cr;
  real_T c2_currentTime;
  real_T c2_d_u;
  real_T c2_e_u;
  real_T c2_f_u;
  real_T c2_q;
  real_T c2_r;
  real_T c2_sr;
  real_T c2_u;
  boolean_T c2_dv1_dirtyOnCpu;
  boolean_T c2_rEQ0;
  if (c2_b_iterationCounter == 0.0) {
    c2_y = NULL;
    sf_mex_assign(&c2_y, sf_mex_create("y", &c2_b_thetaPlotHandle, 0, 0U, 0U, 0U,
      0), false);
    c2_b_y = NULL;
    sf_mex_assign(&c2_b_y, sf_mex_create("y", c2_cv, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_u = rtInf;
    c2_c_y = NULL;
    sf_mex_assign(&c2_c_y, sf_mex_create("y", &c2_u, 0, 0U, 0U, 0U, 0), false);
    c2_d_y = NULL;
    sf_mex_assign(&c2_d_y, sf_mex_create("y", c2_cv1, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_b_u = rtInf;
    c2_e_y = NULL;
    sf_mex_assign(&c2_e_y, sf_mex_create("y", &c2_b_u, 0, 0U, 0U, 0U, 0), false);
    c2_set(chartInstance, c2_y, c2_b_y, c2_c_y, c2_d_y, c2_e_y);
    c2_f_y = NULL;
    sf_mex_assign(&c2_f_y, sf_mex_create("y", &c2_b_alphaPlotHandle, 0, 0U, 0U,
      0U, 0), false);
    c2_g_y = NULL;
    sf_mex_assign(&c2_g_y, sf_mex_create("y", c2_cv, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_c_u = rtInf;
    c2_h_y = NULL;
    sf_mex_assign(&c2_h_y, sf_mex_create("y", &c2_c_u, 0, 0U, 0U, 0U, 0), false);
    c2_i_y = NULL;
    sf_mex_assign(&c2_i_y, sf_mex_create("y", c2_cv1, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_d_u = rtInf;
    c2_k_y = NULL;
    sf_mex_assign(&c2_k_y, sf_mex_create("y", &c2_d_u, 0, 0U, 0U, 0U, 0), false);
    c2_b_set(chartInstance, c2_f_y, c2_g_y, c2_h_y, c2_i_y, c2_k_y);
  }

  if (muDoubleScalarIsNaN(c2_b_iterationCounter) || muDoubleScalarIsInf
      (c2_b_iterationCounter)) {
    c2_r = rtNaN;
  } else if (c2_b_iterationCounter == 0.0) {
    c2_r = 0.0;
  } else {
    c2_r = muDoubleScalarRem(c2_b_iterationCounter, 0.005);
    c2_rEQ0 = (c2_r == 0.0);
    if (!c2_rEQ0) {
      c2_q = muDoubleScalarAbs(c2_b_iterationCounter / 0.005);
      c2_rEQ0 = !(muDoubleScalarAbs(c2_q - muDoubleScalarFloor(c2_q + 0.5)) >
                  2.2204460492503131E-16 * c2_q);
    }

    if (c2_rEQ0) {
      c2_r = 0.0;
    } else {
      if (c2_b_iterationCounter < 0.0) {
        c2_r += 0.005;
      }
    }
  }

  if (c2_r == 0.0) {
    c2_currentTime = c2_b_signals[2];
    c2_cr = muDoubleScalarCos(c2_b_signals[0]);
    c2_sr = muDoubleScalarSin(c2_b_signals[0]);
    c2_YAW = -c2_b_signals[1] * 5.0;
    c2_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (muDoubleScalarSin(c2_YAW), muDoubleScalarCos(c2_YAW), muDoubleScalarSin
       (c2_b_signals[1]), muDoubleScalarCos(c2_b_signals[1]),
       *chartInstance->c2_gpu_BaseToDrive, *chartInstance->c2_gpu_BaseToArm);
    hipMemcpy(chartInstance->c2_gpu_dv1, &c2_b_dv1[0], 4UL,
               hipMemcpyHostToDevice);
    c2_dv1_dirtyOnCpu = false;
    hipMemcpy(chartInstance->c2_gpu_dv, &c2_dv[0], 32UL, hipMemcpyHostToDevice);
    c2_eML_blk_kernel_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c2_gpu_dv1, *chartInstance->c2_gpu_dv,
       *chartInstance->c2_gpu_BaseToDrive, *chartInstance->c2_gpu_BaseToArm);
    c2_alpha1 = 1.0;
    c2_beta1 = 0.0;
    c2_eML_blk_kernel_kernel3<<<dim3(173U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*chartInstance->c2_gpu_armPart_vertices_G_frame);
    hipMemcpy(chartInstance->c2_b_gpu_alpha1, &c2_alpha1, 8UL,
               hipMemcpyHostToDevice);
    hipMemcpy(chartInstance->c2_gpu_armPartVertices, &c2_b_armPartVertices[0],
               705792UL, hipMemcpyHostToDevice);
    hipMemcpy(chartInstance->c2_b_gpu_beta1, &c2_beta1, 8UL,
               hipMemcpyHostToDevice);
    hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 22056, 4,
                (double *)chartInstance->c2_b_gpu_alpha1, (double *)
                &(*chartInstance->c2_gpu_BaseToArm)[0], 4, (double *)
                &(*chartInstance->c2_gpu_armPartVertices)[0], 4, (double *)
                chartInstance->c2_b_gpu_beta1, (double *)
                &(*chartInstance->c2_gpu_armPart_vertices_G_frame)[0], 4);
    c2_eML_blk_kernel_kernel4<<<dim3(44U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*chartInstance->c2_c_gpu_Zt,
       *chartInstance->c2_b_gpu_armPart_vertices_G_frame,
       *chartInstance->c2_c_gpu_Yt,
       *chartInstance->c2_d_gpu_armPart_vertices_G_frame,
       *chartInstance->c2_gpu_Xt,
       *chartInstance->c2_gpu_armPart_vertices_G_frame,
       *chartInstance->c2_c_gpu_armPart_vertices_G_frame);
    c2_j_y = NULL;
    sf_mex_assign(&c2_j_y, sf_mex_create("y", &c2_b_armPartPatchHandle, 0, 0U,
      0U, 0U, 0), false);
    c2_l_y = NULL;
    sf_mex_assign(&c2_l_y, sf_mex_create("y", c2_cv, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_m_y = NULL;
    hipMemcpy(&chartInstance->c2_Xt[0], chartInstance->c2_gpu_Xt, 176448UL,
               hipMemcpyDeviceToHost);
    sf_mex_assign(&c2_m_y, sf_mex_create("y", chartInstance->c2_Xt, 0, 0U, 1U,
      0U, 2, 3, 7352), false);
    c2_n_y = NULL;
    sf_mex_assign(&c2_n_y, sf_mex_create("y", c2_cv1, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_o_y = NULL;
    hipMemcpy(&chartInstance->c2_Yt[0], chartInstance->c2_c_gpu_Yt, 176448UL,
               hipMemcpyDeviceToHost);
    sf_mex_assign(&c2_o_y, sf_mex_create("y", chartInstance->c2_Yt, 0, 0U, 1U,
      0U, 2, 3, 7352), false);
    c2_p_y = NULL;
    sf_mex_assign(&c2_p_y, sf_mex_create("y", c2_cv2, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_q_y = NULL;
    hipMemcpy(&chartInstance->c2_Zt[0], chartInstance->c2_c_gpu_Zt, 176448UL,
               hipMemcpyDeviceToHost);
    sf_mex_assign(&c2_q_y, sf_mex_create("y", chartInstance->c2_Zt, 0, 0U, 1U,
      0U, 2, 3, 7352), false);
    c2_c_set(chartInstance, c2_j_y, c2_l_y, c2_m_y, c2_n_y, c2_o_y, c2_p_y,
             c2_q_y);
    c2_b_dv[0] = 1.0;
    c2_b_dv[4] = 0.0 * c2_sr - 0.0 * c2_cr;
    hipMemcpy(chartInstance->c2_b_gpu_dv, &c2_b_dv[0], 128UL,
               hipMemcpyHostToDevice);
    c2_eML_blk_kernel_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(c2_sr,
      c2_cr, *chartInstance->c2_b_gpu_dv);
    if (c2_dv1_dirtyOnCpu) {
      hipMemcpy(chartInstance->c2_gpu_dv1, &c2_b_dv1[0], 4UL,
                 hipMemcpyHostToDevice);
    }

    c2_eML_blk_kernel_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c2_gpu_dv1, *chartInstance->c2_b_gpu_dv);
    c2_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c2_b_gpu_dv, *chartInstance->c2_gpu_BaseToArm,
       *chartInstance->c2_gpu_y);
    c2_b_alpha1 = 1.0;
    c2_b_beta1 = 0.0;
    c2_eML_blk_kernel_kernel8<<<dim3(51U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*chartInstance->c2_b_gpu_pendPart_vertices_G_frame);
    hipMemcpy(chartInstance->c2_c_gpu_alpha1, &c2_b_alpha1, 8UL,
               hipMemcpyHostToDevice);
    hipMemcpy(chartInstance->c2_gpu_pendPartVertices, &c2_b_pendPartVertices[0],
               208512UL, hipMemcpyHostToDevice);
    hipMemcpy(chartInstance->c2_gpu_beta1, &c2_b_beta1, 8UL,
               hipMemcpyHostToDevice);
    hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 6516, 4,
                (double *)chartInstance->c2_c_gpu_alpha1, (double *)
                &(*chartInstance->c2_gpu_y)[0], 4, (double *)
                &(*chartInstance->c2_gpu_pendPartVertices)[0], 4, (double *)
                chartInstance->c2_gpu_beta1, (double *)
                &(*chartInstance->c2_b_gpu_pendPart_vertices_G_frame)[0], 4);
    c2_eML_blk_kernel_kernel9<<<dim3(13U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*chartInstance->c2_b_gpu_Zt,
       *chartInstance->c2_c_gpu_pendPart_vertices_G_frame,
       *chartInstance->c2_gpu_Yt,
       *chartInstance->c2_d_gpu_pendPart_vertices_G_frame,
       *chartInstance->c2_b_gpu_Xt,
       *chartInstance->c2_b_gpu_pendPart_vertices_G_frame,
       *chartInstance->c2_gpu_pendPart_vertices_G_frame);
    c2_r_y = NULL;
    sf_mex_assign(&c2_r_y, sf_mex_create("y", &c2_b_pendPartPatchHandle, 0, 0U,
      0U, 0U, 0), false);
    c2_s_y = NULL;
    sf_mex_assign(&c2_s_y, sf_mex_create("y", c2_cv, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_t_y = NULL;
    hipMemcpy(&chartInstance->c2_b_Xt[0], chartInstance->c2_b_gpu_Xt, 52128UL,
               hipMemcpyDeviceToHost);
    sf_mex_assign(&c2_t_y, sf_mex_create("y", chartInstance->c2_b_Xt, 0, 0U, 1U,
      0U, 2, 3, 2172), false);
    c2_u_y = NULL;
    sf_mex_assign(&c2_u_y, sf_mex_create("y", c2_cv1, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_v_y = NULL;
    hipMemcpy(&c2_b_Yt[0], chartInstance->c2_gpu_Yt, 52128UL,
               hipMemcpyDeviceToHost);
    sf_mex_assign(&c2_v_y, sf_mex_create("y", c2_b_Yt, 0, 0U, 1U, 0U, 2, 3, 2172),
                  false);
    c2_w_y = NULL;
    sf_mex_assign(&c2_w_y, sf_mex_create("y", c2_cv2, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_x_y = NULL;
    hipMemcpy(&c2_b_Zt[0], chartInstance->c2_b_gpu_Zt, 52128UL,
               hipMemcpyDeviceToHost);
    sf_mex_assign(&c2_x_y, sf_mex_create("y", c2_b_Zt, 0, 0U, 1U, 0U, 2, 3, 2172),
                  false);
    c2_d_set(chartInstance, c2_r_y, c2_s_y, c2_t_y, c2_u_y, c2_v_y, c2_w_y,
             c2_x_y);
    c2_c_alpha1 = 1.0;
    c2_c_beta1 = 0.0;
    c2_eML_blk_kernel_kernel10<<<dim3(30U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*chartInstance->c2_gpu_drivePart_vertices_G_frame);
    hipMemcpy(chartInstance->c2_gpu_alpha1, &c2_c_alpha1, 8UL,
               hipMemcpyHostToDevice);
    hipMemcpy(chartInstance->c2_gpu_drivePartVertices, &c2_b_drivePartVertices
               [0], 119808UL, hipMemcpyHostToDevice);
    hipMemcpy(chartInstance->c2_c_gpu_beta1, &c2_c_beta1, 8UL,
               hipMemcpyHostToDevice);
    hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 3744, 4,
                (double *)chartInstance->c2_gpu_alpha1, (double *)
                &(*chartInstance->c2_gpu_BaseToDrive)[0], 4, (double *)
                &(*chartInstance->c2_gpu_drivePartVertices)[0], 4, (double *)
                chartInstance->c2_c_gpu_beta1, (double *)
                &(*chartInstance->c2_gpu_drivePart_vertices_G_frame)[0], 4);
    c2_eML_blk_kernel_kernel11<<<dim3(8U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*chartInstance->c2_gpu_Zt,
       *chartInstance->c2_d_gpu_drivePart_vertices_G_frame,
       *chartInstance->c2_b_gpu_Yt,
       *chartInstance->c2_b_gpu_drivePart_vertices_G_frame,
       *chartInstance->c2_c_gpu_Xt,
       *chartInstance->c2_gpu_drivePart_vertices_G_frame,
       *chartInstance->c2_c_gpu_drivePart_vertices_G_frame);
    c2_y_y = NULL;
    sf_mex_assign(&c2_y_y, sf_mex_create("y", &c2_b_drivePartPatchHandle, 0, 0U,
      0U, 0U, 0), false);
    c2_ab_y = NULL;
    sf_mex_assign(&c2_ab_y, sf_mex_create("y", c2_cv, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_bb_y = NULL;
    hipMemcpy(&c2_c_Xt[0], chartInstance->c2_c_gpu_Xt, 29952UL,
               hipMemcpyDeviceToHost);
    sf_mex_assign(&c2_bb_y, sf_mex_create("y", c2_c_Xt, 0, 0U, 1U, 0U, 2, 3,
      1248), false);
    c2_cb_y = NULL;
    sf_mex_assign(&c2_cb_y, sf_mex_create("y", c2_cv1, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_db_y = NULL;
    hipMemcpy(&c2_c_Yt[0], chartInstance->c2_b_gpu_Yt, 29952UL,
               hipMemcpyDeviceToHost);
    sf_mex_assign(&c2_db_y, sf_mex_create("y", c2_c_Yt, 0, 0U, 1U, 0U, 2, 3,
      1248), false);
    c2_eb_y = NULL;
    sf_mex_assign(&c2_eb_y, sf_mex_create("y", c2_cv2, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_fb_y = NULL;
    hipMemcpy(&c2_c_Zt[0], chartInstance->c2_gpu_Zt, 29952UL,
               hipMemcpyDeviceToHost);
    sf_mex_assign(&c2_fb_y, sf_mex_create("y", c2_c_Zt, 0, 0U, 1U, 0U, 2, 3,
      1248), false);
    c2_e_set(chartInstance, c2_y_y, c2_ab_y, c2_bb_y, c2_cb_y, c2_db_y, c2_eb_y,
             c2_fb_y);
    c2_gb_y = NULL;
    sf_mex_assign(&c2_gb_y, sf_mex_create("y", &c2_b_thetaPlotHandle, 0, 0U, 0U,
      0U, 0), false);
    c2_hb_y = NULL;
    sf_mex_assign(&c2_hb_y, sf_mex_create("y", c2_cv, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_ib_y = NULL;
    sf_mex_assign(&c2_ib_y, sf_mex_create("y", &c2_currentTime, 0, 0U, 0U, 0U, 0),
                  false);
    sf_mex_assign(&c2_times, c2_horzcat(chartInstance, c2_get(chartInstance,
      c2_gb_y, c2_hb_y), c2_ib_y), false);
    c2_jb_y = NULL;
    sf_mex_assign(&c2_jb_y, sf_mex_create("y", &c2_b_thetaPlotHandle, 0, 0U, 0U,
      0U, 0), false);
    c2_kb_y = NULL;
    sf_mex_assign(&c2_kb_y, sf_mex_create("y", c2_cv1, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_e_u = c2_b_signals[1] * 180.0 / 3.1415926535897931;
    c2_lb_y = NULL;
    sf_mex_assign(&c2_lb_y, sf_mex_create("y", &c2_e_u, 0, 0U, 0U, 0U, 0), false);
    sf_mex_assign(&c2_thetaAngles, c2_b_horzcat(chartInstance, c2_b_get
      (chartInstance, c2_jb_y, c2_kb_y), c2_lb_y), false);
    c2_mb_y = NULL;
    sf_mex_assign(&c2_mb_y, sf_mex_create("y", &c2_b_thetaPlotHandle, 0, 0U, 0U,
      0U, 0), false);
    c2_nb_y = NULL;
    sf_mex_assign(&c2_nb_y, sf_mex_create("y", c2_cv3, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_ob_y = NULL;
    sf_mex_assign(&c2_ob_y, sf_mex_create("y", c2_cv4, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_f_set(chartInstance, c2_mb_y, c2_nb_y, sf_mex_dup(c2_times), c2_ob_y,
             sf_mex_dup(c2_thetaAngles));
    c2_pb_y = NULL;
    sf_mex_assign(&c2_pb_y, sf_mex_create("y", &c2_b_alphaPlotHandle, 0, 0U, 0U,
      0U, 0), false);
    c2_qb_y = NULL;
    sf_mex_assign(&c2_qb_y, sf_mex_create("y", c2_cv, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_rb_y = NULL;
    sf_mex_assign(&c2_rb_y, sf_mex_create("y", &c2_currentTime, 0, 0U, 0U, 0U, 0),
                  false);
    sf_mex_assign(&c2_times, c2_c_horzcat(chartInstance, c2_c_get(chartInstance,
      c2_pb_y, c2_qb_y), c2_rb_y), false);
    c2_sb_y = NULL;
    sf_mex_assign(&c2_sb_y, sf_mex_create("y", &c2_b_alphaPlotHandle, 0, 0U, 0U,
      0U, 0), false);
    c2_tb_y = NULL;
    sf_mex_assign(&c2_tb_y, sf_mex_create("y", c2_cv1, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_f_u = c2_b_signals[0] * 180.0 / 3.1415926535897931;
    c2_ub_y = NULL;
    sf_mex_assign(&c2_ub_y, sf_mex_create("y", &c2_f_u, 0, 0U, 0U, 0U, 0), false);
    sf_mex_assign(&c2_alphaAngles, c2_d_horzcat(chartInstance, c2_d_get
      (chartInstance, c2_sb_y, c2_tb_y), c2_ub_y), false);
    c2_vb_y = NULL;
    sf_mex_assign(&c2_vb_y, sf_mex_create("y", &c2_b_alphaPlotHandle, 0, 0U, 0U,
      0U, 0), false);
    c2_wb_y = NULL;
    sf_mex_assign(&c2_wb_y, sf_mex_create("y", c2_cv3, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_xb_y = NULL;
    sf_mex_assign(&c2_xb_y, sf_mex_create("y", c2_cv4, 10, 0U, 1U, 0U, 2, 1, 5),
                  false);
    c2_g_set(chartInstance, c2_vb_y, c2_wb_y, sf_mex_dup(c2_times), c2_xb_y,
             sf_mex_dup(c2_alphaAngles));
  }

  sf_mex_destroy(&c2_times);
  sf_mex_destroy(&c2_thetaAngles);
  sf_mex_destroy(&c2_alphaAngles);
}

static uint8_T c2_emlrt_marshallIn(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_b_is_active_c2_Non_linear_simulation, const
  char_T *c2_identifier)
{
  emlrtMsgIdentifier c2_thisId;
  uint8_T c2_y;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_y = c2_b_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c2_b_is_active_c2_Non_linear_simulation), &c2_thisId);
  sf_mex_destroy(&c2_b_is_active_c2_Non_linear_simulation);
  return c2_y;
}

static uint8_T c2_b_emlrt_marshallIn(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint8_T c2_b_u;
  uint8_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 3, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static void c2_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                   const mxArray *c2_input0, const mxArray *c2_input1, const
                   mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                   *c2_input4)
{
  sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "set", 0U, 5U, 14, sf_mex_dup
              (c2_input0), 14, sf_mex_dup(c2_input1), 14, sf_mex_dup(c2_input2),
              14, sf_mex_dup(c2_input3), 14, sf_mex_dup(c2_input4));
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  sf_mex_destroy(&c2_input2);
  sf_mex_destroy(&c2_input3);
  sf_mex_destroy(&c2_input4);
}

static void c2_b_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4)
{
  sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "set", 0U, 5U, 14, sf_mex_dup
              (c2_input0), 14, sf_mex_dup(c2_input1), 14, sf_mex_dup(c2_input2),
              14, sf_mex_dup(c2_input3), 14, sf_mex_dup(c2_input4));
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  sf_mex_destroy(&c2_input2);
  sf_mex_destroy(&c2_input3);
  sf_mex_destroy(&c2_input4);
}

static void c2_c_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4, const mxArray *c2_input5, const mxArray
                     *c2_input6)
{
  sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "set", 0U, 7U, 14, sf_mex_dup
              (c2_input0), 14, sf_mex_dup(c2_input1), 14, sf_mex_dup(c2_input2),
              14, sf_mex_dup(c2_input3), 14, sf_mex_dup(c2_input4), 14,
              sf_mex_dup(c2_input5), 14, sf_mex_dup(c2_input6));
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  sf_mex_destroy(&c2_input2);
  sf_mex_destroy(&c2_input3);
  sf_mex_destroy(&c2_input4);
  sf_mex_destroy(&c2_input5);
  sf_mex_destroy(&c2_input6);
}

static void c2_d_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4, const mxArray *c2_input5, const mxArray
                     *c2_input6)
{
  sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "set", 0U, 7U, 14, sf_mex_dup
              (c2_input0), 14, sf_mex_dup(c2_input1), 14, sf_mex_dup(c2_input2),
              14, sf_mex_dup(c2_input3), 14, sf_mex_dup(c2_input4), 14,
              sf_mex_dup(c2_input5), 14, sf_mex_dup(c2_input6));
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  sf_mex_destroy(&c2_input2);
  sf_mex_destroy(&c2_input3);
  sf_mex_destroy(&c2_input4);
  sf_mex_destroy(&c2_input5);
  sf_mex_destroy(&c2_input6);
}

static void c2_e_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4, const mxArray *c2_input5, const mxArray
                     *c2_input6)
{
  sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "set", 0U, 7U, 14, sf_mex_dup
              (c2_input0), 14, sf_mex_dup(c2_input1), 14, sf_mex_dup(c2_input2),
              14, sf_mex_dup(c2_input3), 14, sf_mex_dup(c2_input4), 14,
              sf_mex_dup(c2_input5), 14, sf_mex_dup(c2_input6));
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  sf_mex_destroy(&c2_input2);
  sf_mex_destroy(&c2_input3);
  sf_mex_destroy(&c2_input4);
  sf_mex_destroy(&c2_input5);
  sf_mex_destroy(&c2_input6);
}

static const mxArray *c2_get(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1)
{
  const mxArray *c2_ = NULL;
  c2_ = NULL;
  sf_mex_assign(&c2_, sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "get", 1U,
    2U, 14, sf_mex_dup(c2_input0), 14, sf_mex_dup(c2_input1)), false);
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  return c2_;
}

static const mxArray *c2_horzcat(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1)
{
  const mxArray *c2_ = NULL;
  c2_ = NULL;
  sf_mex_assign(&c2_, sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "horzcat",
    1U, 2U, 14, sf_mex_dup(c2_input0), 14, sf_mex_dup(c2_input1)), false);
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  return c2_;
}

static const mxArray *c2_b_get(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1)
{
  const mxArray *c2_ = NULL;
  c2_ = NULL;
  sf_mex_assign(&c2_, sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "get", 1U,
    2U, 14, sf_mex_dup(c2_input0), 14, sf_mex_dup(c2_input1)), false);
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  return c2_;
}

static const mxArray *c2_b_horzcat(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1)
{
  const mxArray *c2_ = NULL;
  c2_ = NULL;
  sf_mex_assign(&c2_, sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "horzcat",
    1U, 2U, 14, sf_mex_dup(c2_input0), 14, sf_mex_dup(c2_input1)), false);
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  return c2_;
}

static void c2_f_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4)
{
  sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "set", 0U, 5U, 14, sf_mex_dup
              (c2_input0), 14, sf_mex_dup(c2_input1), 14, sf_mex_dup(c2_input2),
              14, sf_mex_dup(c2_input3), 14, sf_mex_dup(c2_input4));
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  sf_mex_destroy(&c2_input2);
  sf_mex_destroy(&c2_input3);
  sf_mex_destroy(&c2_input4);
}

static const mxArray *c2_c_get(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1)
{
  const mxArray *c2_ = NULL;
  c2_ = NULL;
  sf_mex_assign(&c2_, sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "get", 1U,
    2U, 14, sf_mex_dup(c2_input0), 14, sf_mex_dup(c2_input1)), false);
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  return c2_;
}

static const mxArray *c2_c_horzcat(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1)
{
  const mxArray *c2_ = NULL;
  c2_ = NULL;
  sf_mex_assign(&c2_, sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "horzcat",
    1U, 2U, 14, sf_mex_dup(c2_input0), 14, sf_mex_dup(c2_input1)), false);
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  return c2_;
}

static const mxArray *c2_d_get(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1)
{
  const mxArray *c2_ = NULL;
  c2_ = NULL;
  sf_mex_assign(&c2_, sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "get", 1U,
    2U, 14, sf_mex_dup(c2_input0), 14, sf_mex_dup(c2_input1)), false);
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  return c2_;
}

static const mxArray *c2_d_horzcat(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance, const mxArray *c2_input0, const mxArray *c2_input1)
{
  const mxArray *c2_ = NULL;
  c2_ = NULL;
  sf_mex_assign(&c2_, sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "horzcat",
    1U, 2U, 14, sf_mex_dup(c2_input0), 14, sf_mex_dup(c2_input1)), false);
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  return c2_;
}

static void c2_g_set(SFc2_Non_linear_simulationInstanceStruct *chartInstance,
                     const mxArray *c2_input0, const mxArray *c2_input1, const
                     mxArray *c2_input2, const mxArray *c2_input3, const mxArray
                     *c2_input4)
{
  sf_mex_call(chartInstance->c2_fEmlrtCtx, NULL, "set", 0U, 5U, 14, sf_mex_dup
              (c2_input0), 14, sf_mex_dup(c2_input1), 14, sf_mex_dup(c2_input2),
              14, sf_mex_dup(c2_input3), 14, sf_mex_dup(c2_input4));
  sf_mex_destroy(&c2_input0);
  sf_mex_destroy(&c2_input1);
  sf_mex_destroy(&c2_input2);
  sf_mex_destroy(&c2_input3);
  sf_mex_destroy(&c2_input4);
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel1(const
  real_T c2_sy, const real_T c2_cy, const real_T c2_b_sy, const real_T c2_b_cy,
  real_T c2_BaseToDrive[16], real_T c2_BaseToArm[16])
{
  int32_T c2_tmpIdx;
  c2_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c2_tmpIdx < 1) {
    c2_BaseToArm[0] = c2_b_cy;
    c2_BaseToArm[4] = c2_b_cy * 0.0 * 0.0 - c2_b_sy;
    c2_BaseToArm[8] = c2_b_cy * 0.0 + c2_b_sy * 0.0;
    c2_BaseToArm[12] = 0.0;
    c2_BaseToArm[1] = c2_b_sy;
    c2_BaseToArm[5] = c2_b_sy * 0.0 * 0.0 + c2_b_cy;
    c2_BaseToArm[9] = c2_b_sy * 0.0 - c2_b_cy * 0.0;
    c2_BaseToArm[13] = 0.0;
    c2_BaseToDrive[0] = c2_cy;
    c2_BaseToDrive[4] = c2_cy * 0.0 * 0.0 - c2_sy;
    c2_BaseToDrive[8] = c2_cy * 0.0 + c2_sy * 0.0;
    c2_BaseToDrive[12] = 0.0;
    c2_BaseToDrive[1] = c2_sy;
    c2_BaseToDrive[5] = c2_sy * 0.0 * 0.0 + c2_cy;
    c2_BaseToDrive[9] = c2_sy * 0.0 - c2_cy * 0.0;
    c2_BaseToDrive[13] = 0.036;
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel2(const
  int8_T c2_b_dv1[4], const real_T c2_dv[4], real_T c2_BaseToDrive[16], real_T
  c2_BaseToArm[16])
{
  int32_T c2_i;
  c2_i = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i < 4) {
    c2_BaseToArm[(c2_i << 2) + 2] = c2_dv[c2_i];
    c2_BaseToArm[(c2_i << 2) + 3] = (real_T)c2_b_dv1[c2_i];
    c2_BaseToDrive[(c2_i << 2) + 2] = c2_dv[c2_i];
    c2_BaseToDrive[(c2_i << 2) + 3] = (real_T)c2_b_dv1[c2_i];
  }
}

static __global__ __launch_bounds__(512, 1) void c2_eML_blk_kernel_kernel3
  (real_T c2_armPart_vertices_G_frame[88224])
{
  int32_T c2_i1;
  c2_i1 = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i1 < 88224) {
    c2_armPart_vertices_G_frame[c2_i1] = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void c2_eML_blk_kernel_kernel4
  (real_T c2_b_Zt[22056], real_T c2_armPart_vertices_G_frame[22056], real_T
   c2_b_Yt[22056], real_T c2_b_armPart_vertices_G_frame[22056], real_T c2_c_Xt
   [22056], real_T c2_c_armPart_vertices_G_frame[88224], real_T
   c2_d_armPart_vertices_G_frame[22056])
{
  int32_T c2_i2;
  c2_i2 = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i2 < 22056) {
    c2_d_armPart_vertices_G_frame[c2_i2] = c2_c_armPart_vertices_G_frame[c2_i2 <<
      2];
    c2_c_Xt[c2_i2] = c2_d_armPart_vertices_G_frame[c2_i2];
    c2_b_armPart_vertices_G_frame[c2_i2] = c2_c_armPart_vertices_G_frame[(c2_i2 <<
      2) + 1];
    c2_b_Yt[c2_i2] = c2_b_armPart_vertices_G_frame[c2_i2];
    c2_armPart_vertices_G_frame[c2_i2] = c2_c_armPart_vertices_G_frame[(c2_i2 <<
      2) + 2];
    c2_b_Zt[c2_i2] = c2_armPart_vertices_G_frame[c2_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel5(const
  real_T c2_sr, const real_T c2_cr, real_T c2_dv[16])
{
  int32_T c2_tmpIdx;
  c2_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c2_tmpIdx < 1) {
    c2_dv[8] = 0.0 * c2_cr + 0.0 * c2_sr;
    c2_dv[12] = 0.142;
    c2_dv[1] = 0.0;
    c2_dv[5] = 0.0 * c2_sr + c2_cr;
    c2_dv[9] = 0.0 * c2_cr - c2_sr;
    c2_dv[13] = 0.0;
    c2_dv[2] = -0.0;
    c2_dv[6] = c2_sr;
    c2_dv[10] = c2_cr;
    c2_dv[14] = 0.05;
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel6(const
  int8_T c2_b_dv1[4], real_T c2_dv[16])
{
  int32_T c2_i3;
  c2_i3 = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i3 < 4) {
    c2_dv[(c2_i3 << 2) + 3] = (real_T)c2_b_dv1[c2_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel7(const
  real_T c2_dv[16], real_T c2_BaseToArm[16], real_T c2_y[16])
{
  uint64_T c2_threadId;
  int32_T c2_i4;
  int32_T c2_i5;
  int32_T c2_i7;
  c2_threadId = mwGetGlobalThreadIndex();
  c2_i7 = (int32_T)(c2_threadId % 4UL);
  c2_i4 = (int32_T)((c2_threadId - (uint64_T)c2_i7) / 4UL);
  if ((c2_i4 < 4) && (c2_i7 < 4)) {
    c2_y[c2_i4 + (c2_i7 << 2)] = 0.0;
    for (c2_i5 = 0; c2_i5 < 4; c2_i5++) {
      c2_y[c2_i4 + (c2_i7 << 2)] += c2_BaseToArm[c2_i4 + (c2_i5 << 2)] *
        c2_dv[c2_i5 + (c2_i7 << 2)];
    }
  }
}

static __global__ __launch_bounds__(512, 1) void c2_eML_blk_kernel_kernel8
  (real_T c2_pendPart_vertices_G_frame[26064])
{
  int32_T c2_i6;
  c2_i6 = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i6 < 26064) {
    c2_pendPart_vertices_G_frame[c2_i6] = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void c2_eML_blk_kernel_kernel9
  (real_T c2_b_Zt[6516], real_T c2_pendPart_vertices_G_frame[6516], real_T
   c2_b_Yt[6516], real_T c2_b_pendPart_vertices_G_frame[6516], real_T c2_c_Xt
   [6516], real_T c2_c_pendPart_vertices_G_frame[26064], real_T
   c2_d_pendPart_vertices_G_frame[6516])
{
  int32_T c2_i8;
  c2_i8 = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i8 < 6516) {
    c2_d_pendPart_vertices_G_frame[c2_i8] = c2_c_pendPart_vertices_G_frame[c2_i8
      << 2];
    c2_c_Xt[c2_i8] = c2_d_pendPart_vertices_G_frame[c2_i8];
    c2_b_pendPart_vertices_G_frame[c2_i8] = c2_c_pendPart_vertices_G_frame
      [(c2_i8 << 2) + 1];
    c2_b_Yt[c2_i8] = c2_b_pendPart_vertices_G_frame[c2_i8];
    c2_pendPart_vertices_G_frame[c2_i8] = c2_c_pendPart_vertices_G_frame[(c2_i8 <<
      2) + 2];
    c2_b_Zt[c2_i8] = c2_pendPart_vertices_G_frame[c2_i8];
  }
}

static __global__ __launch_bounds__(512, 1) void c2_eML_blk_kernel_kernel10
  (real_T c2_drivePart_vertices_G_frame[14976])
{
  int32_T c2_i9;
  c2_i9 = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i9 < 14976) {
    c2_drivePart_vertices_G_frame[c2_i9] = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void c2_eML_blk_kernel_kernel11
  (real_T c2_b_Zt[3744], real_T c2_drivePart_vertices_G_frame[3744], real_T
   c2_b_Yt[3744], real_T c2_b_drivePart_vertices_G_frame[3744], real_T c2_c_Xt
   [3744], real_T c2_c_drivePart_vertices_G_frame[14976], real_T
   c2_d_drivePart_vertices_G_frame[3744])
{
  int32_T c2_i10;
  c2_i10 = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i10 < 3744) {
    c2_d_drivePart_vertices_G_frame[c2_i10] =
      c2_c_drivePart_vertices_G_frame[c2_i10 << 2];
    c2_c_Xt[c2_i10] = c2_d_drivePart_vertices_G_frame[c2_i10];
    c2_b_drivePart_vertices_G_frame[c2_i10] = c2_c_drivePart_vertices_G_frame
      [(c2_i10 << 2) + 1];
    c2_b_Yt[c2_i10] = c2_b_drivePart_vertices_G_frame[c2_i10];
    c2_drivePart_vertices_G_frame[c2_i10] = c2_c_drivePart_vertices_G_frame
      [(c2_i10 << 2) + 2];
    c2_b_Zt[c2_i10] = c2_drivePart_vertices_G_frame[c2_i10];
  }
}

static void init_dsm_address_info(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc2_Non_linear_simulationInstanceStruct
  *chartInstance)
{
  chartInstance->c2_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c2_signals = (real_T (*)[3])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c2_iterationCounter = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c2_armPartPatchHandle = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c2_pendPartPatchHandle = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c2_drivePartPatchHandle = (real_T *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 4);
  chartInstance->c2_thetaPlotHandle = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 5);
  chartInstance->c2_alphaPlotHandle = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 6);
  chartInstance->c2_frameRate = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 7);
  chartInstance->c2_armPartVertices = (real_T (*)[88224])
    ssGetInputPortSignal_wrapper(chartInstance->S, 8);
  chartInstance->c2_drivePartVertices = (real_T (*)[14976])
    ssGetInputPortSignal_wrapper(chartInstance->S, 9);
  chartInstance->c2_pendPartVertices = (real_T (*)[26064])
    ssGetInputPortSignal_wrapper(chartInstance->S, 10);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c2_Non_linear_simulation_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1378966184U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(132574520U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3765136740U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2332890783U);
}

mxArray *sf_c2_Non_linear_simulation_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,1);
  mxSetCell(mxcell3p, 0, mxCreateString("coder.internal.blas.BLASApi"));
  return(mxcell3p);
}

mxArray *sf_c2_Non_linear_simulation_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("late");
  mxArray *hiddenFallbackReason = mxCreateString("ir_function_calls");
  mxArray *incompatibleSymbol = mxCreateString("gpublascheck");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c2_Non_linear_simulation_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c2_Non_linear_simulation(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiB2YGRgYAPSHEDMxAABrFA+IxKGiLPAxRWAuKSyIBUkXlyU7JkCpPMSc8H8xNI"
    "Kz7y0fLD5FgwI89kImM8JFYcABQfK9EP8F4GknwWLfiUk/QJQfmZxfGJySWZZanyyUbxffl58Tm"
    "ZeamJRfHFmbmlOYklmfh7CfBAAAF9lGV0="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c2_Non_linear_simulation_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQgXoM7hvXpFBsvvSFLvcxE";
}

static void sf_opaque_initialize_c2_Non_linear_simulation(void *chartInstanceVar)
{
  initialize_params_c2_Non_linear_simulation
    ((SFc2_Non_linear_simulationInstanceStruct*) chartInstanceVar);
  initialize_c2_Non_linear_simulation((SFc2_Non_linear_simulationInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c2_Non_linear_simulation(void *chartInstanceVar)
{
  enable_c2_Non_linear_simulation((SFc2_Non_linear_simulationInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c2_Non_linear_simulation(void *chartInstanceVar)
{
  disable_c2_Non_linear_simulation((SFc2_Non_linear_simulationInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c2_Non_linear_simulation(void *chartInstanceVar)
{
  sf_gateway_c2_Non_linear_simulation((SFc2_Non_linear_simulationInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c2_Non_linear_simulation(SimStruct*
  S)
{
  return get_sim_state_c2_Non_linear_simulation
    ((SFc2_Non_linear_simulationInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c2_Non_linear_simulation(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c2_Non_linear_simulation
    ((SFc2_Non_linear_simulationInstanceStruct*)sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c2_Non_linear_simulation(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc2_Non_linear_simulationInstanceStruct*) chartInstanceVar)
      ->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_Non_linear_simulation_optimization_info();
    }

    mdl_cleanup_runtime_resources_c2_Non_linear_simulation
      ((SFc2_Non_linear_simulationInstanceStruct*) chartInstanceVar);
    ((SFc2_Non_linear_simulationInstanceStruct*) chartInstanceVar)->
      ~SFc2_Non_linear_simulationInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c2_Non_linear_simulation(void *chartInstanceVar)
{
  mdl_start_c2_Non_linear_simulation((SFc2_Non_linear_simulationInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c2_Non_linear_simulation(void
  *chartInstanceVar)
{
  mdl_terminate_c2_Non_linear_simulation
    ((SFc2_Non_linear_simulationInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc2_Non_linear_simulation
    ((SFc2_Non_linear_simulationInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c2_Non_linear_simulation(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c2_Non_linear_simulation
      ((SFc2_Non_linear_simulationInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc2_Non_linear_simulation
      ((SFc2_Non_linear_simulationInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c2_Non_linear_simulation_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV82P00YUd1YLKqKg5VSpqgTlAhIX1As9IciH1aCEXXAW9hbNjl/iUcYz7nx4syfuIHHnjHr",
    "puX9MT5w58ifwxnY+cOyEdAXaIkbyesf+vZ/f95t4jW7fw3UVr/fXPO8i3n/Aa8fL14Vi31i68u",
    "e73q1i/w8KCRsfEEVi7a1dgsTwFLTk1jApumIkK2FMjECBoIhNpDJ1bJrFljMx8a2gjk8/jxiNg",
    "khaHjZRloT7gp8iW2LNAfK0mQJqfIDQREraceRzMp5rrMxJKwI60TZeZ4IGE9jEqaX7lhuWcOhM",
    "gXaFNgQ11gvdAkMMtMy01kxnqQ5mQBknnBFRaW1EdAAJOtjAYRLi331r0KgyjEZEmSZEJAXdY5O",
    "MUwooczKNL46ZIEYqRngn5i0nuKrbAUd9+jIEvsYhqFtTAZkkkglTH//AR0s7ghxzaMOxHdezBf",
    "CndcF/xuAEVK3fRi2ZgiJj2Be1H80c0plm0ZpnySrMsBieEfWQYvw0hLXZi5mjA4JxggFK1MEgM",
    "7KrB4ql6N5aNht3XWZuKhkb58HWm2AZWyeFdVGYs/lUtAjnuhY2kEkPUuAZa5sYsh6Ws1bjtGbh",
    "QKKDXXrXV4MVDANfwFpShKwyXGkJkPWdx9hYPkVSq42MW5i87V5v9fUqrCsMqBGhUNUFFGEa0Ge",
    "Ze+vZQqZd7BGIWplMvSpwniGbUJ4eWdE+kWqCPlnTRBYmuIjWAmM9xlhiJRxqLJp1MBfLTThKaA",
    "ShazCMQx/LBrEVPtGutT3EukuZOW2DpoolFVG1WHXYhjouoU4TOBQTIU+Er2QcFD0+jwJ2BuzhM",
    "cZgkNWYoEjFtMF2wRafDwEwKYkSTIyb2ObUqY9KVkbMzb273mLu/fgZc28mV77fXuJpVPB4S3eH",
    "f7CEv7TzKf5y6bs7s2eLVclzpfTd3RKPw+3h9fb1u86dR3+/fPXz3l9vpi9+LfuhrE9jRZ9G9r+",
    "T+9DY7txwtdj/MmvQ84JLV/LcYf9Y0mu3gv+nJf69Yq+fjI9k/16UHiV+U6dp4PdSOu3k/tpS39",
    "nzG25SYHZmea1oNywONG5PbD7mHf/vS/pe3MB/qXierxsPziaf++tog79ulvx1M5unQ+KqFIb0t",
    "+FjKYZYc1hBw+yIRVzFrub7f82XbeW8ryz3f9Hzu9yXj/vnzIPLZ5wj513urPZtOxfPG/7umn7q",
    "lfB759iOs55XvjT+X2+7uXy92N+f/8RoRYyHFafN4jWeCUdVb7+RPN0G/xGG5b5i",
    ""
  };

  static char newstr [1265] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c2_Non_linear_simulation(SimStruct *S)
{
  const char* newstr = sf_c2_Non_linear_simulation_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1172213411U));
  ssSetChecksum1(S,(2326415915U));
  ssSetChecksum2(S,(2769296011U));
  ssSetChecksum3(S,(562002073U));
}

static void mdlRTW_c2_Non_linear_simulation(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c2_Non_linear_simulation(SimStruct *S)
{
  SFc2_Non_linear_simulationInstanceStruct *chartInstance;
  chartInstance = (SFc2_Non_linear_simulationInstanceStruct *)utMalloc(sizeof
    (SFc2_Non_linear_simulationInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc2_Non_linear_simulationInstanceStruct));
  chartInstance = new (chartInstance) SFc2_Non_linear_simulationInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c2_Non_linear_simulation;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c2_Non_linear_simulation;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c2_Non_linear_simulation;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c2_Non_linear_simulation;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c2_Non_linear_simulation;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c2_Non_linear_simulation;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c2_Non_linear_simulation;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c2_Non_linear_simulation;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c2_Non_linear_simulation;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c2_Non_linear_simulation;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c2_Non_linear_simulation;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c2_Non_linear_simulation;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c2_JITStateAnimation,
    chartInstance->c2_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c2_Non_linear_simulation(chartInstance);
}

void c2_Non_linear_simulation_method_dispatcher(SimStruct *S, int_T method, void
  *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c2_Non_linear_simulation(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c2_Non_linear_simulation(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c2_Non_linear_simulation(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c2_Non_linear_simulation_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
